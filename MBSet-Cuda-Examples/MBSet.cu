#include "hip/hip_runtime.h"
/* 
 * File:   MBSet.cu
 * 
 * Created on June 24, 2012
 * 
 * Purpose:  This program displays Mandelbrot set using the GPU via CUDA and
 * OpenGL immediate mode.
 * 
 */

#include <iostream>
#include <stack>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include "Complex.cu"
#include <string.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <time.h>
#include <vector>
#include <cmath>
#include <GL/freeglut.h>
#include <GL/glut.h>
#include <GL/glext.h>
#include <GL/gl.h>
#include <GL/glu.h>

using namespace std;

// Size of window in pixels, both width and height
#define WINDOW_DIM 512
#define THREADSPERBLOCK 16
#define NUMBLOCKS (WINDOW_DIM*WINDOW_DIM/THREADSPERBLOCK)

// all variables
// Min and max complex plane values
Complex  minC(-2.0, -1.2);
Complex  maxC( 1.0, 1.8);
int maxIt = 2000;     // Max iterations for the set computations
float aspect;
int win_id;
int N;
bool button_down = false;
float x_down, y_down, x_up, y_up, x_current, y_current;
bool selectionDone = false;
Complex new_minC(minC.r, minC.i);
Complex new_maxC(maxC.r, maxC.i);
bool drawBox_flag = false;

vector<float> color1;
vector<float> color2;
vector<float> color3;
int MBdata [262144];
vector<float> minCdata_r;
vector<float> minCdata_i;
vector<float> maxCdata_r;
vector<float> maxCdata_i;
int *dev_MBdata;  // device copy

// function declarations
void display(void);
void init();
void reshape(int w, int h);
void mouse(int button, int state, int x, int y);
void motion(int x, int y);
void keyboard(unsigned char c, int x, int y);
void thread_create();
void drawBox();
void InitializeColors();

void InitializeColors()
{
srand48(time(NULL));
  for(int i = 0; i < 2000; i++)
  { 
    color1.push_back(drand48());
    color2.push_back(drand48());
    color3.push_back(drand48());
  }  
}

void thread_create()
{
  hipMalloc( (void**)&dev_MBdata, WINDOW_DIM*WINDOW_DIM*sizeof(int));
  generateMB<<< NUMBLOCKS, THREADSPERBLOCK >>>(new_minC, new_maxC, dev_MBdata, maxIt);
}


void display(void)
{
 // Your OpenGL display code here
  
  glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);
  glLoadIdentity();


  int MBdata_cell = 0;
  for (int i = 0; i < winH; i++)
  {
    for (int j = 0; j < winW; j++)
    {
      MBdata_cell = MBdata[i * winW + j];
      if(MBdata_cell != maxIt && MBdata_cell > 5)
      {
        //not a part of Mandelbrot Set
        glColor3f(color1[MBdata_cell], color2[MBdata_cell], color3[MBdata_cell]); //random color
        glBegin(GL_POINTS); //starts drawing of points
        glVertex2f(j, winW - i);
        glEnd();//end drawing of points       
      }
      else if (MBdata_cell <= 5)
      {
        //not a part of Mandelbrot Set
        glColor3f(1.0, 1.0, 1.0); //white color
        glBegin(GL_POINTS); //starts drawing of points
        glVertex2f(j, winW - i);
        glEnd();//end drawing of points         
      }
      else
      {
        glColor3f(0.0, 0.0, 0.0); //black color
        glBegin(GL_POINTS); //starts drawing of points
        glVertex2f(j, winW - i);
        glEnd();//end drawing of points           
      } 

      // if(j < 5)
      // {
      //  glColor3f(1.0, 0.0, 0.0); //black color
      //  glBegin(GL_POINTS); //starts drawing of points
      //  glVertex2f(j, winW - i);
      //  glEnd();//end drawing of points         
      // }  
      // if(i < 5)
      // {
      //  glColor3f(0.0, 0.0, 1.0); //black color
      //  glBegin(GL_POINTS); //starts drawing of points
      //  glVertex2f(j, winW - i);
      //  glEnd();//end drawing of points 
      // }  
    }
  }

  if(drawBox_flag)
  {
    drawBox();
  }

  glutSwapBuffers(); // If double buffering  
}

void drawBox()
{
  drawBox_flag = false;
  glColor3f(1.0, 0.0, 0.0); //red color
  glLineWidth(2.0);
  glBegin(GL_LINE_LOOP); //starts drawing of points
  glVertex2f(x_down, winW - y_down);
  glVertex2f(x_down, winW - y_current);
  glVertex2f(x_current, winW - y_current);
  glVertex2f(x_current, winW - y_down);
  glEnd();

}

void reshape(int w, int h)
{ // Your OpenGL window reshape code here
  glMatrixMode(GL_PROJECTION);
  glLoadIdentity();
  int width = h * aspect;
  int left = (w - width) / 2; 
  glViewport(left, 0, width, (GLsizei)h);
  glOrtho(0.0, winW, 0.0, winH, -winW, winW);
  glMatrixMode(GL_MODELVIEW); 
  glutPostRedisplay();

}

void mouse(int button, int state, int x, int y)
{ // Your mouse click processing here
  // state == 0 means pressed, state != 0 means released
  // Note that the x and y coordinates passed in are in
  // PIXELS, with y = 0 at the top.
  // state == 0 means pressed, state != 0 means released
  // Note that the x and y coordinates passed in are in
  // PIXELS, with y = 0 at the top.

  if(button == GLUT_LEFT_BUTTON && state == GLUT_DOWN)
  {
    selectionDone = false;
    button_down = true;
    x_down = x;
    y_down = y;

    // cout<<x_down<<" "<<y_down<<"\t";
  }
  if((button_down == true) && (button == GLUT_LEFT_BUTTON && state == GLUT_UP))
  {
    button_down = false;
    x_up = x;
    y_up = y;
    selectionDone = true;
    // cout<<x_up<<" "<<y_up<<endl;
  } 
  if(selectionDone == true)
  {

    // first click top left
    if ((x_down < x_up) && (y_down < y_up))
    {
      float delta = min((x_up - x_down), (y_up - y_down));
      x_up = x_down + delta;
      y_up = y_down + delta;
    }
    // first click top right
    else if ((x_down > x_up) && (y_down < y_up))
    {
      float delta = min((x_down - x_up), (y_up - y_down));
      x_up = x_down - delta;
      y_up = y_down + delta;
      int temp_x_down = x_up;
      int temp_x_up = x_down;
      x_down = temp_x_down;
      x_up = temp_x_up;
    }
    // first click bottom right
    else if ((x_down > x_up) && (y_down > y_up))
    {
      float delta = min((x_down - x_up), (y_down - y_up));
      x_up = x_down - delta;
      y_up = y_down - delta;
      int temp_x_down = x_up;
      int temp_x_up = x_down;
      int temp_y_down = y_up;
      int temp_y_up = y_down;
      x_down =  temp_x_down;
      x_up = temp_x_up;
      y_down = temp_y_down;
      y_up = temp_y_up;
    }
    // first click bottom left
    else if((x_down < x_up) && (y_down > y_up))
    {
      float delta = min((x_up - x_down), (y_down - y_up));
      x_up = x_down + delta;
      y_up = y_down - delta;
      int temp_y_down = y_up;
      int temp_y_up = y_down;
      y_up = temp_y_up;
      y_down = temp_y_down;
    }

    // x_down = 0;
    // y_down = 0;
    // x_up = 511;
    // y_up = 511;

    float x_down_scaled = new_minC.r + (float)x_down*(new_maxC.r - new_minC.r)/(float)winW;
    float x_up_scaled = new_minC.r + (float)x_up*(new_maxC.r - new_minC.r)/(float)winW;
    float y_down_scaled = new_minC.i + (float)y_down*(new_maxC.i - new_minC.i)/(float)winW;
    float y_up_scaled = new_minC.i + (float)y_up*(new_maxC.i - new_minC.i)/(float)winW;

    // cout<<x_down<<"\t"<<x_down_scaled<<"  "<<endl;
    // cout<<y_down<<"\t"<<y_down_scaled<<"  "<<endl;
    // cout<<x_up<<"\t"<<x_up_scaled<<"  "<<endl;
    // cout<<y_up<<"\t"<<y_up_scaled<<"  "<<endl;

    minCdata_r.push_back(new_minC.r);
    minCdata_i.push_back(new_minC.i);

    maxCdata_r.push_back(new_maxC.r);
    maxCdata_i.push_back(new_maxC.i);

    new_minC.r = x_down_scaled;
    new_minC.i = y_down_scaled;
    new_maxC.r = x_up_scaled;
    new_maxC.i = y_up_scaled;

    // cout<<endl<<"final values: "<<new_minC.r<<" "<<new_minC.i<<"\t"<<new_maxC.r<<" "<<new_maxC.i<<endl;
    selectionDone = false;
    drawBox_flag = false;

      thread_create();
        
    display();
    glutPostRedisplay();
  }

}

void motion(int x, int y)
{
  drawBox_flag = false;
  if(button_down)
  {
    x_current = x;
    y_current = y;
    // first click top left
    if ((x_down < x_current) && (y_down < y_current))
    {
      float delta = min((x_current - x_down), (y_current - y_down));
      x_current = x_down + delta;
      y_current = y_down + delta;
    }
    // first click top right
    else if ((x_down > x_current) && (y_down < y_current))
    {
      float delta = min((x_down - x_current), (y_current - y_down));
      x_current = x_down - delta;
      y_current = y_down + delta;
    }
    // first click bottom right
    else if ((x_down > x_up) && (y_down > y_up))
    {
      float delta = min((x_down - x_current), (y_down - y_current));
      x_current = x_down - delta;
      y_current = y_down - delta;
    }
    // first click bottom left
    else if((x_down < x_current) && (y_down > y_current))
    {
      float delta = min((x_current - x_down), (y_down - y_current));
      x_current = x_down + delta;
      y_current = y_down - delta;
    }
    drawBox_flag = true;
    // display();
    glutPostRedisplay();
  }

  
}

void keyboard(unsigned char c, int x, int y)
{
  if (c == 'b')

  {
    if(minCdata_i.size())
    {
      glutPostRedisplay();
      new_minC.r = minCdata_r.back();
      new_minC.i = minCdata_i.back();
      new_maxC.r = maxCdata_r.back();
      new_maxC.i = maxCdata_i.back();
      minCdata_r.pop_back();
      minCdata_i.pop_back();
      maxCdata_r.pop_back();
      maxCdata_i.pop_back();

        thread_create();
      display();
    }
  }

  else if (c == 'e')
  {
    glutDestroyWindow(win_id);
    exit(0);    
  }
  
}

void init()
{ // Your OpenGL initialization code here
  glClearColor(1.0, 1.0, 1.0, 1.0);
  glShadeModel(GL_FLAT);
}


__global__ void generateMB(Complex new_minC, Complex new_maxC, int* dev_MBdata, int maxIt)
{
  int index = threadId.x + blockIdx * blockDim.x;
  int i = index % WINDOW_DIM;
  int j = index / WINDOW_DIM;

  Complex c;  

  float c_r = new_minC.r + (float)j*(new_maxC.r - new_minC.r)/(float)winW;
  float c_i = new_minC.i + (float)i*(new_maxC.i - new_minC.i)/(float)winH;

  c.r = c_r;
  c.i  = c_i;
  Complex z = Complex(c.r, c.i);
  int iter_no = 0;

  while((z.magnitude2() < 4.0) && (iter_no < maxIt))
  {
    z = z * z + c;
    iter_no++;
  }

  dev_MBdata[i * winW + j] = iter_no;
}

int main(int argc, char** argv)
{
  winW = 512;
  winH = 512; 
  N = winW;
  aspect = (float)winW/(float)winH;
  
  // initialize colors
  InitializeColors();

  pthread_barrier_init(&barrier, NULL, nThreads);     // Initializing the barrier

    thread_create();

  // Initialize OpenGL, but only on the "master" thread or process.
  // See the assignment writeup to determine which is "master" 
  // and which is slave.
  // pthread_barrier_wait(&barrier);

  glutInit(&argc, argv);
  glutInitDisplayMode(GLUT_DOUBLE | GLUT_RGB | GLUT_DEPTH);
  glutInitWindowSize(winW, winH);
  glutInitWindowPosition(300, 100);
  
  string title = "Mandelbrot Set";
  win_id = glutCreateWindow(title.c_str());
  init();


  glutDisplayFunc(display);
  glutReshapeFunc(reshape);
  glutKeyboardFunc(keyboard);
  glutMotionFunc(motion);
  glutMouseFunc(mouse);

  glutMainLoop();
  
  return 0;
  
}
